
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ 
void device_hello(){

  //uncomment this line to print only one time (unless you have multiple blocks)
  //if(threadIdx.x==0)
  printf("Hello world! from the device! thread:%d,%d\n",blockIdx.x,threadIdx.x);
  return;
}

int main(void){

  // rather than calling fflush    
  setbuf(stdout, NULL);

  // greet from the host
  printf("Hello world! from the host!\n");

  // launch a kernel with a block of threads to greet from the device
  dim3 blockSize(32,5,1);
  dim3 gridSize(1,1,1);
  // run several variations by playing with the block and grid sizes
  // above -- if you change the y or z dimensions change the print 
  // statement to reflect that.
  device_hello<<<gridSize,blockSize>>>();

  // comment this line out and see what happens
  hipDeviceSynchronize();

  printf("Goodbye world! from the host!\n");

  return 0;
}
