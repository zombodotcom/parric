#include "hip/hip_runtime.h"
#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <math.h>
#include <stdlib.h>

#include <sys/time.h>

#define min(x,y) ( ((x) < (y))? (x) : (y))


/*
 * Note that the innermost loop can be 0<=j<=i to take 
 * advantage of the symmetry in the input matrix for this
 * kernel. This will only compute half of the output C
 * matrix, saving half the computation. You can try to 
 * tile this version by replacing the upper bound on j
 * with <= N. (This is slightly more complicated.)
 *
 */ 
__global__ void kernel_syr2k(int N, int M,
    double C[ N][N],
    double A[ N][M],
    double B[ N][M])
{
  int i, j, k;

  for (i = 0; i < N; i++) {
    for (k = 0; k < M; k++) {
      for (j = 0; j < N; j++) {
        C[i][j] += A[j][k] * B[i][k] + B[j][k] * A[i][k];
      }
    }
  }
}


void init_array(int N, int M,
  double C[N][N],
  double A[N][M],
  double B[N][M])
{
  int i, j;

  for (i = 0; i < N; i++)
    for (j = 0; j < M; j++) {
      A[i][j] = (double) (i*j%N) / N;
      B[i][j] = (double) (i*j%M) / M;
    }
  for (i = 0; i < N; i++)
    for (j = 0; j < N; j++)
      C[i][j] = (double) (i*j%N) / M;
}



void print_array(int N,
   double C[N][N])
{
  int i, j;

  fprintf(stderr, "==BEGIN DUMP_ARRAYS==\n");
  fprintf(stderr, "begin dump: %s", "C");
  for (i = 0; i < N; i++)
    for (j = 0; j < N; j++) {
 if ((i * N + j) % 20 == 0) fprintf (stderr, "\n");
 fprintf (stderr, "%0.2lf ", C[i][j]);
    }
  fprintf(stderr, "\nend   dump: %s\n", "C");
  fprintf(stderr, "==END   DUMP_ARRAYS==\n");
}


int main(int argc, char** argv)
{
  dim3 NumBlocks, threadsPerBlock;

  int N;
  int M;

  struct timeval t_start;
  struct timeval t_end;
  double etime;

  double* C;
  double* A;
  double* B;

  if (argc < 3) {
    printf("usage ./syr2k N M\n");
    return 0;
  }

  N = atoi(argv[1]);
  M = atoi(argv[2]);
  
 

  C = (double*)malloc(N*N * sizeof(double));
  A = (double*)malloc(N*M * sizeof(double));
  B = (double*)malloc(N*M * sizeof(double));

  init_array (N, M, *((double(*)[N][N])C), *((double(*)[N][M])A), *((double(*)[N][M])B));

  gettimeofday (&t_start, NULL);

  kernel_syr2k <<<NumBlocks, threadsPerBlock >>>
	   (N, M, *((double(*)[N][N])C), *((double(*)[N][M])A), *((double(*)[N][M])B));

  gettimeofday (&t_end, NULL);

  etime = t_end.tv_sec - t_start.tv_sec + 
        (t_end.tv_usec - t_start.tv_usec) * 1.0e-6;

  print_array(N, *((double(*)[N][N])C));

  printf("execution time=%lf\n", etime);

  free(C);
  free(A);
  free(B);

  return 0;
}
